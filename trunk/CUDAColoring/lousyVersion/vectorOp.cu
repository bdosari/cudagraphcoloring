#include "hip/hip_runtime.h"
#include "vectorOp.h"


//Vector addition: C = a + b
__global__ void vectorAdd(float *cD, float *aD, float *bD, int N){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N)
		cD[i] = aD[i] + bD[i];	
} 




//Vector addition: C = a * b
__global__ void vectorMul(float *cD, float *aD, float *bD, int N){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N)
		cD[i] = aD[i] * bD[i];	
} 




extern "C" __host__ void vectorAddMulPrep(float *c, float *a, float *b, int N, int choice){
	float *cD, *aD, *bD;

	// Allocating memory on device
	hipMalloc((void**)&cD, N * sizeof(float));
	hipMalloc((void**)&aD, N * sizeof(float));
	hipMalloc((void**)&bD, N * sizeof(float));

	
	// transfer data - a & b - from host(CPU) to device(GPU)
	hipMemcpy(aD, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bD, b, N * sizeof(float), hipMemcpyHostToDevice);

	
	//invoke function on GPU
	if (choice == 0)
		//launchAddKernel(cD, aD, bD, N);
		vectorAdd<<<ceil(N/(float)BLOCK_SIZE),BLOCK_SIZE>>>(cD, aD, bD, N);
	else
		//launchMulKernel(cD, aD, bD, N);
		vectorMul<<<ceil(N/(float)BLOCK_SIZE),BLOCK_SIZE>>>(cD, aD, bD, N);


	// transfer result of addition - c - from device(GPU) to host(CPU)
	hipMemcpy(c, cD, N * sizeof(float), hipMemcpyDeviceToHost);


	// free memory
	hipFree(cD);
	hipFree(aD);
	hipFree(bD);
}
