#include "hip/hip_runtime.h"
#include "graphColoring.h"
using namespace std;

//Author: Pascal

__global__ void colorGraph(int *adjacencyMatrixD, int *colors, int size, int maxDegree){
	int i, j, start, end;
	int subGraphSize, numColors = 0;
	
	subGraphSize = size/(gridDim.x * blockDim.x);
	start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	end = start + subGraphSize;

	int degreeArray[100];

	for(i=start; i<end; i++)
	{
		for(j=0; j<=maxDegree; j++)
			degreeArray[j] = j+1;
	
		for(j=start; j<end; j++){
			if(i==j)
			    continue;

			if(adjacencyMatrixD[i*size + j] == 1)
				if(colors[j] != 0)
					degreeArray[colors[j]-1] = 0;
		}	   

		for(j=0; j<=maxDegree; j++)
			if(degreeArray[j] != 0){
				colors[i] = degreeArray[j];
				break;
			}

		if(colors[i] > numColors)
			numColors = colors[i];
		
	
	}
}


//Author: Pascal
extern "C"
__host__ void subGraphColoring(int *adjacencyMatrix, int *graphColors, int maxDegree)
{
	int *adjacencyMatrixD, *colorsD;
        
	hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));

	hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(GRIDSIZE);
	dim3 dimBlock(BLOCKSIZE);

	colorGraph<<<dimGrid, dimBlock>>>(adjacencyMatrixD, colorsD, GRAPHSIZE, maxDegree);
	
	hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(adjacencyMatrixD);
	hipFree(colorsD);

}


// Author :Peihong
__global__ void detectConflicts(int *adjacencyMatrixD, int *colors, int *conflictD, int size){
	int i, j, start, end;
	int subGraphSize, numColors = 0;
	
	subGraphSize = size/(gridDim.x * blockDim.x);
	start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	end = start + subGraphSize;

	if(end > size) end = size;

        for(i=start; i<end; i++)
	{
		for(j=end; j < size; j++)
	        {
			if(adjacencyMatrixD[i*size + j] == 1 && (colors[i] == colors[j]))
			{
				conflictD[min(i,j)] = 1;	
			}
		}
	}

}

//Author: Peihong
__global__ void detectConflicts(int *adjacencyMatrixD, int *boundaryListD, int *colors, int *conflictD, int size, int boundarySize){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i;
	if(idx < boundarySize){
		i = boundaryListD[idx];
		conflictD[idx] = 0;
		for(int j= i+1; j < size; j++)
		{
			if(adjacencyMatrixD[j*size + i] == 1 && (colors[i] == colors[j]))
			{
				//conflictD[idx] = min(i,j)+1;	
				conflictD[idx] = i+1;	
			}		
		}
	}
}


/*__global__ void detectConflicts(int *adjacencyMatrixD, int *boundaryListD, int *colors, int *conflictD, int size, int boundarySize){
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i < size){

		for(int idx= 0; idx < boundarySize; idx++)
		{
			int j =  boundaryListD[idx];
			conflictD[idx] = 0;
			if( adjacencyMatrixD[j*size + i] == 1 && (colors[i] == colors[j]))
			{
				conflictD[idx] = j+1;	
			}		
		}
	}
	__syncthreads();
}*/

// Author:Peihong
extern "C"
void colorConfilctDetection(int *adjacencyMatrix, int *boundaryList, int *graphColors, int *conflict, int boundarySize)
{
	//cout<<"enter conflict detection wrapper"<<endl;	
	int *adjacencyMatrixD, *colorsD, *conflictD, *boundaryListD;
	
        
	hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&conflictD, boundarySize*sizeof(int));
	hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));

	hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);

        int gridsize = ceil((float)boundarySize/(float)SUBSIZE_BOUNDARY);
	int blocksize = SUBSIZE_BOUNDARY;
	//int gridsize = ceil((float)GRAPHSIZE/(float)BLOCKSIZE);
	//int blocksize = BLOCKSIZE;

	dim3 dimGrid(gridsize);
	dim3 dimBlock(blocksize);
	
        //cout<<"call conflict detection kernel"<<endl;
	detectConflicts<<<dimGrid, dimBlock>>>(adjacencyMatrixD, boundaryListD, colorsD, conflictD, GRAPHSIZE, boundarySize);
	//detectConflicts<<<GRIDSIZE, BLOCKSIZE>>>(adjacencyMatrixD, colorsD, conflictD, GRAPHSIZE);

        //cout<<"call kernel complete"<<endl;
	hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(conflict, conflictD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(adjacencyMatrixD);
	hipFree(colorsD);
	hipFree(conflictD);
	hipFree(boundaryListD);
}














