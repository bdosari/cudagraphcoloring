#include "hip/hip_runtime.h"
#include "graphColoring.h"



//----------------------- SDO improved -----------------------//
//
// Author: Shusen & Pascal
// returns the degree of that node
int __device__ degree(int vertex, int *degreeList){
	return degreeList[vertex];
}



// Author: Shusen & Pascal
// saturation of a vertex
int __device__ saturation(int vertex, int *adjacencyList, int *graphColors, int maxDegree, int start, int end){
	int saturation = 0;	
	int colors[256];
	for (int j=0; j<256; j++)
		colors[j] = 0;
	
	
	for (int i=0; i<maxDegree; i++){
		if (adjacencyList[vertex*maxDegree + i] < start)
			continue;
		
		if (adjacencyList[vertex*maxDegree + i] > end)
			break;
		
		if (adjacencyList[vertex*maxDegree + i] != -1)
			//colors[ graphColors[vertex] ] = 1;			// at each colored set the array to 1
			colors[ graphColors[adjacencyList[vertex*maxDegree + i]] ] = 1;			// at each colored set the array to 1
		else
			break;
	}
	
	
	for (int i=1; i<maxDegree+1; i++)					// count the number of 1's but skip uncolored
		if (colors[i] == 1)
			saturation++;
	
	return saturation;
}




// Author: Shusen & Pascal
// colors the vertex with the min possible color
int __device__ color(int vertex, int *adjacencyList, int *graphColors, int maxDegree, int numColored, int start, int end, int disp){
	int colors[256];
	for (int j=0; j<246; j++)
		colors[j] = 0;
	
	
	if (graphColors[vertex] == 0)
		numColored++;
	
	for (int i=0; i<maxDegree; i++){						// set the index of the color to 1	
		// Limits color checking to subgraph
		/*
		 if (adjacencyList[vertex*maxDegree + i] < start)
		 continue;
		 
		 if (adjacencyList[vertex*maxDegree + i] > end)
		 break;
		 */
		
		if (adjacencyList[vertex*maxDegree + i] != -1)
			colors[  graphColors[  adjacencyList[vertex*maxDegree + i]  ]  ] = 1;
		else 
			break;
	}
	
	
	for (int i=1; i<maxDegree+1; i++)					// nodes still equal to 0 are unassigned
		if (colors[i] != 1){
			if (disp == 0){
				graphColors[vertex] = i;
				break;
			}
			else
				disp--;
		}
	
	return numColored;
}





// Author: Shusen & Pascal
// does the coloring
__global__ void colorGraph_SDO(int *adjacencyList, int *graphColors, int *degreeList, int sizeGraph, int maxDegree, 
								int *startPartitionListD, int *endPartitionListD, int *randomListD)
{
	int start, end, partitionIndex;
	int subGraphSize, numColored = 0;
	int satDegree, max, index;
	int randomCount = 0;
	
	//subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
	//start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	subGraphSize = end - start;
	
	while (numColored < subGraphSize){
		randomCount++;
		randomCount = randomCount%10;
		
		max = -1;
		
		for (int i=start; i<end; i++){
			if (graphColors[i] == 0)			// not colored
			{
				satDegree = saturation(i,adjacencyList,graphColors, maxDegree, start, end);
				
				if (satDegree > max){
					max = satDegree;
					index = i;				
				}
				
				if (satDegree == max){
					if (degree(i,degreeList) > degree(index,degreeList))
						index = i;
				}
				
				
			}
			numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
		}
	}
}






//Author: Pascal
//recolors nodes where we have conflicts
__global__ void conflictSolveSDO(int *adjacencyList, int *conflict, int *graphColors, int *degreeList, 
								int sizeGraph, int maxDegree, int *startPartitionListD, int *endPartitionListD, int *randomListD){
	int start, end, index, partitionIndex;
	int numColored = 0;
	int satDegree, max;
	int randomCount = 0;
	int numOfInitialConflicts = 0;
	
	
	// int subGraphSize;
	//subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
	//start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	
	
	
	// Count the number of conflicts
	for (int i=start; i<end; i++)
		if (graphColors[i] == 0)
			numOfInitialConflicts++;
    
	
    while (numOfInitialConflicts > 0){
        max = -1;
        randomCount++;
		randomCount = randomCount%10;
        
        for (int i=start; i<end; i++){
            if (graphColors[i] == 0)                        // not colored
            {
				satDegree = saturation(i,adjacencyList,graphColors, maxDegree, start, end);
				
                if (satDegree > max){
                    max = satDegree;
                    index = i;                              
                }
				
                if (satDegree == max){
                    if (degree(i,degreeList) > degree(index,degreeList))
                        index = i;
                }
				
				numColored = color(index,adjacencyList,graphColors, maxDegree, numColored, start, end, randomListD[partitionIndex*10 + randomCount]);
				
				if (i == index)
					numOfInitialConflicts--;
            }
        }
    }
}





//----------------------- First Fit Adjacency List -----------------------//
//
// Author: Pascal
// First Fit
__global__ void colorGraph_FF(int *adjacencyListD, int *colors, int size, int maxDegree, int *startPartitionListD, int *endPartitionListD){
	int i, j, start, end, partitionIndex;
	int numColors = 0;
	
	//int subGraphSize;
	//subGraphSize = size/(gridDim.x * blockDim.x);
	//start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	//end = start + subGraphSize;
	
	partitionIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	start = startPartitionListD[partitionIndex];
	end = endPartitionListD[partitionIndex];
	
	
	int degreeArray[100];
	for (i=start; i<end; i++)
	{
		for(j=0; j<maxDegree; j++)
			degreeArray[j] = j+1;
		
		
		for (j=0; j<maxDegree; j++){
			int vertexNeigh = i*maxDegree + j;
			
			if (adjacencyListD[vertexNeigh] != -1){
				if (colors[ adjacencyListD[vertexNeigh] ] != 0)
					degreeArray[ colors[adjacencyListD[vertexNeigh]] -1 ] = 0;
			}
			else
				break;
		}
		
		
		for(j=0; j<maxDegree; j++)
			if(degreeArray[j] != 0){
				colors[i] = degreeArray[j];
				break;
			}
		
		if(colors[i] > numColors)
			numColors = colors[i];		
	}
}



//----------------------- Detects conflicts -----------------------//
//
// Author: Peihong
// each thread deals with 1 vertex from boundary list
// 		set the conflicted color to 0
// 		set its value in the conflict list to point to the node
__global__ void conflictsDetection(int *adjacentListD, int *boundaryListD, int *colors, int *conflictD, long size, int boundarySize, int maxDegree){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int nodeFrom, nodeTo;
	
	
	if (idx < boundarySize){
		nodeFrom = boundaryListD[idx];
		

		for (int i=0; i<maxDegree; i++)
		{
			nodeTo = adjacentListD[nodeFrom*maxDegree + i];
			
			if (nodeTo == -1)
				break;
			
			if (nodeFrom>=nodeTo && (colors[nodeFrom] == colors[nodeTo]))
			{
				conflictD[idx] = nodeFrom;	
				colors[nodeFrom] = 0;				// added!!!!!!!!
			}		
		}
	}
}




//----------------------- Main -----------------------//

extern "C"
void cudaGraphColoring(int *adjacentList, int *boundaryList, int *graphColors, int *degreeList, int *conflict, int boundarySize, 
						int maxDegree, int graphSize, int passes, int subsizeBoundary, int _gridSize, int _blockSize, 
						int *startPartitionList, int *endPartitionList, int *randomList, int numRand)
{
	int *adjacentListD, *colorsD, *boundaryListD, *degreeListD, *conflictListD, *startPartitionListD, *endPartitionListD, *randomListD;     
	int gridsize = ceil((float)boundarySize/(float)(256));
	int blocksize = 256;
	int *numConflicts;
	
	hipEvent_t start_col, start_confl, stop_col, stop_confl, start_mem, stop_mem;         
    float elapsedTime_memory, elapsedTime_col, elapsedTime_confl; 
	
	
	
	
	//-------------- memory transfer -----------------!
	hipEventCreate(&start_mem); 
    hipEventCreate(&stop_mem); 
    hipEventRecord(start_mem, 0); 
	
	
	hipMalloc((void**)&adjacentListD, graphSize*maxDegree*sizeof(int));
	hipMalloc((void**)&colorsD, graphSize*sizeof(int));
	hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));
	hipMalloc((void**)&degreeListD, graphSize*sizeof(int));
	hipMalloc((void**)&numConflicts, 1*sizeof(int));
	hipMalloc((void**)&conflictListD, boundarySize*sizeof(int));
	hipMalloc((void**)&startPartitionListD, _gridSize*_blockSize*sizeof(int));
	hipMalloc((void**)&endPartitionListD, _gridSize*_blockSize*sizeof(int));
	hipMalloc((void**)&randomListD, numRand*sizeof(int));
	
	
	
	hipMemcpy(adjacentListD, adjacentList, graphSize*maxDegree*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(colorsD, graphColors, graphSize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(degreeListD, degreeList, graphSize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(startPartitionListD, startPartitionList, _gridSize*_blockSize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(endPartitionListD, endPartitionList, _gridSize*_blockSize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(randomListD, randomList, numRand*sizeof(int), hipMemcpyHostToDevice);
	
	
	
	
	hipEventRecord(stop_mem, 0); 
    hipEventSynchronize(stop_mem); 
	
	
	
	
	dim3 dimGrid_col(_gridSize);
	dim3 dimBlock_col(_blockSize);
	
	dim3 dimGrid_confl(gridsize);
	dim3 dimBlock_confl(blocksize);
	
	
	
	
	//-------------- Sequential Graph coloring -----------------!
	hipEventCreate(&start_col); 
    hipEventCreate(&stop_col); 
    hipEventRecord(start_col, 0); 
	
	
	//colorGraph_FF<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, graphSize, maxDegree);				// First Fit
	colorGraph_SDO<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, degreeListD, 
											graphSize, maxDegree, startPartitionListD, endPartitionListD, randomListD);		// SDO improved
	
	
	hipEventRecord(stop_col, 0); 
    hipEventSynchronize(stop_col); 
	
	
	
	
	
	for (int times=1; times<passes; times++){
		//-------------- Conflict resolution -----------------!
		hipEventCreate(&start_confl); 
		hipEventCreate(&stop_confl); 
		hipEventRecord(start_confl, 0); 
		
		hipMemset(conflictListD, -1, boundarySize*sizeof(int));
		conflictsDetection<<<dimGrid_confl, dimBlock_confl>>>(adjacentListD, boundaryListD, colorsD, conflictListD, graphSize, boundarySize, maxDegree);
		
		hipEventRecord(stop_confl, 0); 
		hipEventSynchronize(stop_confl); 
		
		
		hipEventCreate(&start_col); 
		hipEventCreate(&stop_col); 
		hipEventRecord(start_col, 0); 
		
		conflictSolveSDO<<<dimGrid_col, dimBlock_col>>>(adjacentListD, conflictListD, colorsD, degreeListD, graphSize, 
														maxDegree, startPartitionListD, endPartitionListD, randomListD);
		
		hipEventRecord(stop_col, 0); 
		hipEventSynchronize(stop_col); 
	}
	
	
	hipEventCreate(&start_confl); 
    hipEventCreate(&stop_confl); 
    hipEventRecord(start_confl, 0); 
	
	hipMemset(conflictListD, -1, boundarySize*sizeof(int));
	conflictsDetection<<<dimGrid_confl, dimBlock_confl>>>(adjacentListD, boundaryListD, colorsD, conflictListD, graphSize, boundarySize, maxDegree);
	
	hipEventRecord(stop_confl, 0); 
    hipEventSynchronize(stop_confl); 
	
	
	
	//-------------- Cleanup -----------------!
	hipMemcpy(graphColors, colorsD, graphSize*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(conflict, conflictListD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);
	
	
	
	hipEventElapsedTime(&elapsedTime_memory, start_mem, stop_mem); 
	hipEventElapsedTime(&elapsedTime_col, start_col, stop_col); 
	hipEventElapsedTime(&elapsedTime_confl, start_confl, stop_confl); 
	
	cout << endl << "GPU timings ~ Memory transfer: " << elapsedTime_memory  << " ms     Coloring: " 
	<< elapsedTime_col << " ms    Conflict: " << elapsedTime_confl << " ms" << endl; 
	
	
	hipFree(adjacentListD);
	hipFree(colorsD);
	hipFree(boundaryListD);
	hipFree(degreeListD);
	hipFree(numConflicts);
	hipFree(conflictListD);
	hipFree(startPartitionListD);
	hipFree(endPartitionListD);
	hipFree(randomListD);
}

