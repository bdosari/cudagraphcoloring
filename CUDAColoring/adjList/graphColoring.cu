#include "hip/hip_runtime.h"
#include "graphColoring.h"
using namespace std;

//Author: Shusen

// returns the degree of that node
int __device__ degree(int vertex, int *degreeList){
	return degreeList[vertex];
}

int __device__ saturation(int vertex, int *adjacencyList, int *graphColors, int maxDegree){
	int saturation = 0;
	int colors[100];

	//memset(colors, 0, (maxDegree+1)*sizeof(int));		// initialize array


	for (int i=0; i<maxDegree; i++){
		if (adjacencyList[vertex*maxDegree + i] != -1)
			colors[ graphColors[vertex] ] = 1;			// at each colored set the array to 1
		else
			break;
	}


	for (int i=1; i<maxDegree+1; i++)					// count the number of 1's but skip uncolored
		if (colors[i] == 1)
			saturation++;

	return saturation;
}



// colors the vertex with the min possible color
int __device__ color(int vertex, int *adjacencyList, int *graphColors, int maxDegree, int numColored){
	int colors[100];
	
	if (graphColors[vertex] == 0)
		numColored++;
	
	for (int i=0; i<maxDegree; i++)						// set the index of the color to 1
		if (adjacencyList[vertex*maxDegree + i] != -1)
			colors[  graphColors[  adjacencyList[vertex*maxDegree + i]  ]  ] = 1;
		else {
			break;
		}

	

	for (int i=1; i<maxDegree+1; i++)					// nodes still equal to 0 are unassigned
		if (colors[i] != 1){
			graphColors[vertex] = i;
			break;
		}
	
	return numColored;
}

__global__ void colorGraphAdjL_complex(int *adjacencyList, int *graphColors, int *degreeList, int sizeGraph, int maxDegree)
{
	int i, j, start, end;
	int subGraphSize, numColored = 0;
	int satDegree, max, index;
	
	subGraphSize = sizeGraph/(gridDim.x * blockDim.x);
	start = (sizeGraph/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	end = start + subGraphSize;

	while (numColored < subGraphSize){
		max = -1;
		
		for (int i=start; i<end; i++){
			if (graphColors[i] == 0)			// not colored
			{
				satDegree = saturation(i,adjacencyList,graphColors, maxDegree);

				if (satDegree > max){
					max = satDegree;
					index = i;				
				}

				if (satDegree == max){
					if (degree(i,degreeList) > degree(index,degreeList))
						index = i;
				}
			}

			numColored = color(index,adjacencyList,graphColors, maxDegree, numColored);
			//iterations++;
		}
	}


}


//Author: Pascal
__global__ void colorGraphAdjL(int *adjacencyListD, int *colors, int size, int maxDegree){
	int i, j, start, end;
	int subGraphSize, numColors = 0;
	
	subGraphSize = size/(gridDim.x * blockDim.x);
	start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	end = start + subGraphSize;
	

	int degreeArray[100];
	for(i=start; i<end; i++)
	{
		for(j=0; j<maxDegree; j++)
			degreeArray[j] = j+1;


		for (j=0; j<maxDegree; j++){
			int vertexNeigh = i*maxDegree + j;

			if (adjacencyListD[vertexNeigh] != -1){
				if (colors[ adjacencyListD[vertexNeigh] ] != 0)
					degreeArray[ colors[adjacencyListD[vertexNeigh]] -1 ] = 0;
			}
			else
				break;
		}
		

		for(j=0; j<maxDegree; j++)
			if(degreeArray[j] != 0){
				colors[i] = degreeArray[j];
				break;
			}
		
		if(colors[i] > numColors)
			numColors = colors[i];		
	}
}



//Author: Pascal
__global__ void colorGraph(int *adjacencyMatrixD, int *colors, int size, int maxDegree){
	int i, j, start, end;
	int subGraphSize, numColors = 0;
	
	subGraphSize = size/(gridDim.x * blockDim.x);
	start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	end = start + subGraphSize;
	
	int degreeArray[100];
	
	for(i=start; i<end; i++)
	{
		for(j=0; j<=maxDegree; j++)
			degreeArray[j] = j+1;
		
		for(j=start; j<end; j++){
			if(i==j)
			    continue;
			
			if(adjacencyMatrixD[i*size + j] == 1)
				if(colors[j] != 0)
					degreeArray[colors[j]-1] = 0;
		}	   
		
		for(j=0; j<=maxDegree; j++)
			if(degreeArray[j] != 0){
				colors[i] = degreeArray[j];
				break;
			}
		
		if(colors[i] > numColors)
			numColors = colors[i];		
	}
}


//Author: Pascal
extern "C"
__host__ void subGraphColoring(int *adjacencyMatrix, int *graphColors, int maxDegree)
{
	int *adjacencyMatrixD, *colorsD;
	
	hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));
	
	hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
	
	dim3 dimGrid(GRIDSIZE);
	dim3 dimBlock(BLOCKSIZE);
	
	colorGraph<<<dimGrid, dimBlock>>>(adjacencyMatrixD, colorsD, GRAPHSIZE, maxDegree);
	
	hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(adjacencyMatrixD);
	hipFree(colorsD);
	
}


// Author :Peihong
__global__ void detectConflicts(int *adjacencyMatrixD, int *colors, int *conflictD, int size){
	int i, j, start, end;
	int subGraphSize, numColors = 0;
	
	subGraphSize = size/(gridDim.x * blockDim.x);
	start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
	end = start + subGraphSize;
	
	if(end > size) end = size;
	
	for(i=start; i<end; i++)
	{
		for(j=end; j < size; j++)
		{
			if(adjacencyMatrixD[i*size + j] == 1 && (colors[i] == colors[j]))
			{
				conflictD[min(i,j)] = 1;	
			}
		}
	}
	
}

//Author: Peihong
__global__ void detectConflicts(int *adjacencyMatrixD, int *boundaryListD, int *colors, int *conflictD, int size, int boundarySize){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i;
	if(idx < boundarySize){
		i = boundaryListD[idx];
		conflictD[idx] = 0;
		for(int j= i+1; j < size; j++)
		{
			if(adjacencyMatrixD[j*size + i] == 1 && (colors[i] == colors[j]))
			{
				//conflictD[idx] = min(i,j)+1;	
				conflictD[idx] = i+1;	
			}		
		}
	}
}


/*__global__ void detectConflicts(int *adjacencyMatrixD, int *boundaryListD, int *colors, int *conflictD, int size, int boundarySize){
 int i = blockIdx.x*blockDim.x + threadIdx.x;
 
 if(i < size){
 
 for(int idx= 0; idx < boundarySize; idx++)
 {
 int j =  boundaryListD[idx];
 conflictD[idx] = 0;
 if( adjacencyMatrixD[j*size + i] == 1 && (colors[i] == colors[j]))
 {
 conflictD[idx] = j+1;	
 }		
 }
 }
 __syncthreads();
 }*/



// Author:Peihong
extern "C"
void colorConfilctDetection(int *adjacencyMatrix, int *boundaryList, int *graphColors, int *conflict, int boundarySize)
{
	int *adjacencyMatrixD, *colorsD, *conflictD, *boundaryListD;
	
	/**
	 hipEvent_t start, stop;         
	 float elapsedTimeCPU; 
	 
	 hipEventCreate(&start); 
     hipEventCreate(&stop); 
     hipEventRecord(start, 0);       
	 /**/      
	
	hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&conflictD, boundarySize*sizeof(int));
	hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));
	
	hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);
	
	
	/**
	 hipEventRecord(stop, 0); 
	 hipEventSynchronize(stop); 
	 hipEventElapsedTime(&elapsedTimeCPU, start, stop); 
	 
	 cout << "GPU time: " << elapsedTimeCPU << endl; 
	 /**/
	
	
    int gridsize = ceil((float)boundarySize/(float)SUBSIZE_BOUNDARY);
	int blocksize = SUBSIZE_BOUNDARY;
	
	
	dim3 dimGrid(gridsize);
	dim3 dimBlock(blocksize);
	
	detectConflicts<<<dimGrid, dimBlock>>>(adjacencyMatrixD, boundaryListD, colorsD, conflictD, GRAPHSIZE, boundarySize);
	
	hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(conflict, conflictD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(adjacencyMatrixD);
	hipFree(colorsD);
	hipFree(conflictD);
	hipFree(boundaryListD);
}




// Author:Peihong & Pascal
// Description: Merging of colorConfilctDetection & subGraphColoring
//				to save on data transfer time
extern "C"
void colorAndConflict(int *adjacencyMatrix, int *boundaryList, int *graphColors, int *conflict, int boundarySize, int maxDegree)
{
	int *adjacencyMatrixD, *colorsD, *conflictD, *boundaryListD;     
	int gridsize = ceil((float)boundarySize/(float)SUBSIZE_BOUNDARY);
	int blocksize = SUBSIZE_BOUNDARY;
	
	hipEvent_t start_col, start_confl, stop_col, stop_confl, start_mem, stop_mem;         
    float elapsedTime_memory, elapsedTime_col, elapsedTime_confl; 
	
	
	// memory transfer
	hipEventCreate(&start_mem); 
    hipEventCreate(&stop_mem); 
    hipEventRecord(start_mem, 0); 
	
	hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&conflictD, boundarySize*sizeof(int));
	hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));
	
	hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);
	
	
	hipEventRecord(stop_mem, 0); 
    hipEventSynchronize(stop_mem); 
	
	
	
	
	dim3 dimGrid_col(GRIDSIZE);
	dim3 dimBlock_col(BLOCKSIZE);
	
	dim3 dimGrid_confl(gridsize);
	dim3 dimBlock_confl(blocksize);
	
	
	// Graph coloring
	hipEventCreate(&start_col); 
    hipEventCreate(&stop_col); 
    hipEventRecord(start_col, 0); 
	
	colorGraph<<<dimGrid_col, dimBlock_col>>>(adjacencyMatrixD, colorsD, GRAPHSIZE, maxDegree);
	
	
	hipEventRecord(stop_col, 0); 
    hipEventSynchronize(stop_col); 

	
	// Conflict resolution
	hipEventCreate(&start_confl); 
    hipEventCreate(&stop_confl); 
    hipEventRecord(start_confl, 0); 
	
	detectConflicts<<<dimGrid_confl, dimBlock_confl>>>(adjacencyMatrixD, boundaryListD, colorsD, conflictD, GRAPHSIZE, boundarySize);
	
	hipEventRecord(stop_confl, 0); 
    hipEventSynchronize(stop_confl); 
	
	hipEventElapsedTime(&elapsedTime_memory, start_mem, stop_mem); 
	hipEventElapsedTime(&elapsedTime_col, start_col, stop_col); 
	hipEventElapsedTime(&elapsedTime_confl, start_confl, stop_confl); 
	cout << "GPU time ~ Memory: " << elapsedTime_memory  << "  Color: " << elapsedTime_col << "  Conflict: " << elapsedTime_confl << endl; 
	
	
	hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(conflict, conflictD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(adjacencyMatrixD);
	hipFree(colorsD);
	hipFree(conflictD);
	hipFree(boundaryListD);
}






//Author: Peihong
__global__ void conflictsDetection(int *adjacentListD, int *boundaryListD, int *colors, int *conflictD, int size, int boundarySize, int maxDegree){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i, j;
	if(idx < boundarySize){
		i = boundaryListD[idx];
		conflictD[idx] = 0;
		for(int k= 0; k < maxDegree; k++)
		{
			j = adjacentListD[i*maxDegree + k];
			if(j>=0 && (colors[i] == colors[j]))
			{
				//conflictD[idx] = min(i,j)+1;	
				conflictD[idx] = i+1;	
			}		
		}
	}
}

// Author:Peihong & Pascal
// Description: Merging of colorConfilctDetection & subGraphColoring
//				to save on data transfer time
extern "C"
void cudaGraphColoring(int *adjacentList, int *boundaryList, int *graphColors, int *conflict, int boundarySize, int maxDegree)
{
	int *adjacentListD, *colorsD, *conflictD, *boundaryListD;     
	int gridsize = ceil((float)boundarySize/(float)SUBSIZE_BOUNDARY);
	int blocksize = SUBSIZE_BOUNDARY;
	
	hipEvent_t start_col, start_confl, stop_col, stop_confl, start_mem, stop_mem;         
    float elapsedTime_memory, elapsedTime_col, elapsedTime_confl; 
	
/*
	// Adj list display
	for (int i=0; i<10; i++){
		for (int j=0; j<maxDegree; j++){
			cout << adjacentList[i*maxDegree + j] << " ";
		}
		cout << endl;
	}
*/

	cout << "Max deg: " << maxDegree << endl;


	// memory transfer
	hipEventCreate(&start_mem); 
    hipEventCreate(&stop_mem); 
    hipEventRecord(start_mem, 0); 
	
	hipMalloc((void**)&adjacentListD, GRAPHSIZE*maxDegree*sizeof(int));
	hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&conflictD, boundarySize*sizeof(int));
	hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));
	
	hipMemcpy(adjacentListD, adjacentList, GRAPHSIZE*maxDegree*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);
	
	
	hipEventRecord(stop_mem, 0); 
    hipEventSynchronize(stop_mem); 
	
	
	dim3 dimGrid_col(GRIDSIZE);
	dim3 dimBlock_col(BLOCKSIZE);
	
	dim3 dimGrid_confl(gridsize);
	dim3 dimBlock_confl(blocksize);
	
	
	// Graph coloring
	hipEventCreate(&start_col); 
        hipEventCreate(&stop_col); 
        hipEventRecord(start_col, 0); 
	
	//colorGraph<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, GRAPHSIZE, maxDegree);
	colorGraphAdjL<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, GRAPHSIZE, maxDegree);
	
	
	hipEventRecord(stop_col, 0); 
        hipEventSynchronize(stop_col); 

	
	// Conflict resolution
	hipEventCreate(&start_confl); 
        hipEventCreate(&stop_confl); 
        hipEventRecord(start_confl, 0); 
	
	conflictsDetection<<<dimGrid_confl, dimBlock_confl>>>(adjacentListD, boundaryListD, colorsD, conflictD, GRAPHSIZE, boundarySize, maxDegree);
	
	hipEventRecord(stop_confl, 0); 
    hipEventSynchronize(stop_confl); 
	
	hipEventElapsedTime(&elapsedTime_memory, start_mem, stop_mem); 
	hipEventElapsedTime(&elapsedTime_col, start_col, stop_col); 
	hipEventElapsedTime(&elapsedTime_confl, start_confl, stop_confl); 
	cout << "GPU time ~ Memory: " << elapsedTime_memory  << "  Color: " << elapsedTime_col << "  Conflict: " << elapsedTime_confl << endl; 
	
	
	hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(conflict, conflictD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);

	

	hipFree(adjacentListD);
	hipFree(colorsD);
	hipFree(conflictD);
	hipFree(boundaryListD);
}


extern "C"
void cudaGraphColoring_complex(int *adjacentList, int *boundaryList, int *graphColors, int *degreeList, int *conflict, int boundarySize, int maxDegree)
{
	int *adjacentListD, *colorsD, *conflictD, *boundaryListD, *degreeListD;     
	int gridsize = ceil((float)boundarySize/(float)SUBSIZE_BOUNDARY);
	int blocksize = SUBSIZE_BOUNDARY;
	
	hipEvent_t start_col, start_confl, stop_col, stop_confl, start_mem, stop_mem;         
    float elapsedTime_memory, elapsedTime_col, elapsedTime_confl; 
	
/*
	// Adj list display
	for (int i=0; i<10; i++){
		for (int j=0; j<maxDegree; j++){
			cout << adjacentList[i*maxDegree + j] << " ";
		}
		cout << endl;
	}
*/

	cout << "Max deg: " << maxDegree << endl;


	// memory transfer
	hipEventCreate(&start_mem); 
    hipEventCreate(&stop_mem); 
    hipEventRecord(start_mem, 0); 
	
	hipMalloc((void**)&adjacentListD, GRAPHSIZE*maxDegree*sizeof(int));
	hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));
	hipMalloc((void**)&conflictD, boundarySize*sizeof(int));
	hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));
	hipMalloc((void**)&degreeListD, GRAPHSIZE*sizeof(int));
	
	hipMemcpy(adjacentListD, adjacentList, GRAPHSIZE*maxDegree*sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(degreeListD, degreeList, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
	
	
	hipEventRecord(stop_mem, 0); 
    hipEventSynchronize(stop_mem); 
	
	
	dim3 dimGrid_col(GRIDSIZE);
	dim3 dimBlock_col(BLOCKSIZE);
	
	dim3 dimGrid_confl(gridsize);
	dim3 dimBlock_confl(blocksize);
	
	
	// Graph coloring
	hipEventCreate(&start_col); 
        hipEventCreate(&stop_col); 
        hipEventRecord(start_col, 0); 
	
	//colorGraph<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, GRAPHSIZE, maxDegree);
	//colorGraphAdjL<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, GRAPHSIZE, maxDegree);
	colorGraphAdjL_complex<<<dimGrid_col, dimBlock_col>>>(adjacentListD, colorsD, degreeListD,GRAPHSIZE, maxDegree);
	
	
	hipEventRecord(stop_col, 0); 
        hipEventSynchronize(stop_col); 

	
	// Conflict resolution
	hipEventCreate(&start_confl); 
        hipEventCreate(&stop_confl); 
        hipEventRecord(start_confl, 0); 
	
	conflictsDetection<<<dimGrid_confl, dimBlock_confl>>>(adjacentListD, boundaryListD, colorsD, conflictD, GRAPHSIZE, boundarySize, maxDegree);
	
	hipEventRecord(stop_confl, 0); 
    hipEventSynchronize(stop_confl); 
	
	hipEventElapsedTime(&elapsedTime_memory, start_mem, stop_mem); 
	hipEventElapsedTime(&elapsedTime_col, start_col, stop_col); 
	hipEventElapsedTime(&elapsedTime_confl, start_confl, stop_confl); 
	cout << "GPU time ~ Memory: " << elapsedTime_memory  << "  Color: " << elapsedTime_col << "  Conflict: " << elapsedTime_confl << endl; 
	
	
	hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(conflict, conflictD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);

	

	hipFree(adjacentListD);
	hipFree(colorsD);
	hipFree(conflictD);
	hipFree(boundaryListD);
}










