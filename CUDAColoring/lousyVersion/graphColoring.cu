#include "hip/hip_runtime.h"
#include "graphColoring.h"

// adjacencyMatrixD:adjacncy matrix
// colors:			storage for colors
// size: 		 	number of nodes
// subGraphSize: 	number of nodes per subgraph
// maxDegree:	    maximum degree of the graph
__global__ void colorGraph(int *adjacencyMatrixD, int *colors, int size, int subGraphSize, int maxDegree){ 
	int i, j, start, end;
	int numColors = 0; 

	start = (size/gridDim.x * blockIdx.x) + (subGraphSize/blockDim.x * threadIdx.x);	// y position of start of row
	end = start + subGraphSize;

	//int *degreeArray; 
	//degreeArray = new int[maxDegree+1]; 
	int degreeArray[50]; 
	

	for (i=start; i<end; i++) 
	{                
		// initialize degree array: stores colors used
		for (j=0; j<=maxDegree; j++) 
			degreeArray[j] = j+1; 
		
		
		// check the colors 
		for (j=start; j<end; j++){ 
			if (i == j) 
				continue; 
			
			// check connected 
			if (adjacencyMatrixD[i*size + j] == 1) 
				if (colors[j-start] != 0) 
					degreeArray[colors[j-start]-1] = 0;   // set connected spots to 0 
		} 
		

		for (j=0; j<=maxDegree; j++) 
			if (degreeArray[j-start] != 0){ 
				colors[i-start] = degreeArray[j-start]; 
				break; 
			} 
		
		if (colors[(i-start)] > numColors) 
			numColors=colors[(i-start)]; 
	} 

	//delete[] degreeArray;
} 



extern "C" __host__ void subGraphColoring(int *adjacencyMatrix, int *graphColors, int maxDegree)
{
	// partitioning
	int numSub = ceil((float)GRAPHSIZE/(float)SUBSIZE);
	memset(graphColors, 0, GRAPHSIZE*sizeof(int));  


	
	int k, maxColor = 1;
	int *adjacencyMatrixD, *colorsD;

	// Allocating memory on device
	hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE * sizeof(int));
	hipMalloc((void**)&colorsD, 		  GRAPHSIZE * sizeof(int));


	// transfer data to destination [ device(GPU) ] from source [ host(CPU) ]
	hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(colorsD, 			 graphColors,     GRAPHSIZE * sizeof(int),        hipMemcpyHostToDevice);


	// graph coloring
	dim3 dimGrid( ceil(GRAPHSIZE/(float)BLOCKSIZE_Y) );
	dim3 dimBlock( BLOCKSIZE_Y );
	colorGraph<<<dimGrid, dimBlock>>>(adjacencyMatrixD, colorsD, GRAPHSIZE, SUBSIZE, maxDegree);


	// transfer result to destination[ host(CPU) ] from source from [ device(GPU) ]
	hipMemcpy(graphColors, colorsD, GRAPHSIZE * sizeof(int), hipMemcpyDeviceToHost);


	// free memory
	hipFree(adjacencyMatrixD);
	hipFree(colorsD);

	

	//cout<<"partitioned graphColors:"<<endl;	
	printf("Partitioned graph colors: \n"); 
	for (k=0; k<GRAPHSIZE; k++) 
		//cout << graphColors[k] << "  "; 
		printf("%d ", graphColors[k]);
	
	printf("\n"); 
	//cout << endl; 
	//cout<<"number of colors:"<< maxColor << endl;
}
