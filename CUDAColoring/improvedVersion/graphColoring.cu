#include "hip/hip_runtime.h"
#include "graphColoring.h"
using namespace std;

//Author: Pascal

__global__ void colorGraph(int *adjacencyMatrixD, int *colors, int size, int maxDegree){
int i, j, start, end;
int subGraphSize, numColors = 0;

subGraphSize = size/(gridDim.x * blockDim.x);
start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
end = start + subGraphSize;

int degreeArray[100];

for(i=start; i<end; i++)
{
for(j=0; j<=maxDegree; j++)
degreeArray[j] = j+1;

for(j=start; j<end; j++){
if(i==j)
continue;

if(adjacencyMatrixD[i*size + j] == 1)
if(colors[j] != 0)
degreeArray[colors[j]-1] = 0;
}

for(j=0; j<=maxDegree; j++)
if(degreeArray[j] != 0){
colors[i] = degreeArray[j];
break;
}

if(colors[i] > numColors)
numColors = colors[i];
}
}


//Author: Pascal
extern "C"
__host__ void subGraphColoring(int *adjacencyMatrix, int *graphColors, int maxDegree)
{
int *adjacencyMatrixD, *colorsD;

hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE*sizeof(int));
hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));

hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);

dim3 dimGrid(GRIDSIZE);
dim3 dimBlock(BLOCKSIZE);

colorGraph<<<dimGrid, dimBlock>>>(adjacencyMatrixD, colorsD, GRAPHSIZE, maxDegree);

hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
hipFree(adjacencyMatrixD);
hipFree(colorsD);

}


// Author :Peihong
__global__ void detectConflicts(int *adjacencyMatrixD, int *colors, int *conflictD, int size){
int i, j, start, end;
int subGraphSize, numColors = 0;

subGraphSize = size/(gridDim.x * blockDim.x);
start = (size/gridDim.x * blockIdx.x) + (subGraphSize * threadIdx.x);
end = start + subGraphSize;

if(end > size) end = size;

for(i=start; i<end; i++)
{
for(j=end; j < size; j++)
{
if(adjacencyMatrixD[i*size + j] == 1 && (colors[i] == colors[j]))
{
conflictD[min(i,j)] = 1;
}
}
}

}

//Author: Peihong
__global__ void detectConflicts(int *adjacencyMatrixD, int *boundaryListD, int *colors, int *conflictD, int size, int boundarySize){
int idx = blockIdx.x*blockDim.x + threadIdx.x;
int i;
if(idx < boundarySize){
i = boundaryListD[idx];
conflictD[idx] = 0;
for(int j= i+1; j < size; j++)
{
if(adjacencyMatrixD[j*size + i] == 1 && (colors[i] == colors[j]))
{
//conflictD[idx] = min(i,j)+1;
conflictD[idx] = i+1;
}
}
}
}


/*__global__ void detectConflicts(int *adjacencyMatrixD, int *boundaryListD, int *colors, int *conflictD, int size, int boundarySize){
int i = blockIdx.x*blockDim.x + threadIdx.x;

if(i < size){

for(int idx= 0; idx < boundarySize; idx++)
{
int j = boundaryListD[idx];
conflictD[idx] = 0;
if( adjacencyMatrixD[j*size + i] == 1 && (colors[i] == colors[j]))
{
conflictD[idx] = j+1;
}
}
}
__syncthreads();
}*/



// Author:Peihong
extern "C"
void colorConfilctDetection(int *adjacencyMatrix, int *boundaryList, int *graphColors, int *conflict, int boundarySize)
{
int *adjacencyMatrixD, *colorsD, *conflictD, *boundaryListD;

/**
hipEvent_t start, stop;
float elapsedTimeCPU;

hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);
/**/

hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE*sizeof(int));
hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));
hipMalloc((void**)&conflictD, boundarySize*sizeof(int));
hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));

hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);


/**
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTimeCPU, start, stop);

cout << "GPU time: " << elapsedTimeCPU << endl;
/**/


int gridsize = ceil((float)boundarySize/(float)SUBSIZE_BOUNDARY);
int blocksize = SUBSIZE_BOUNDARY;


dim3 dimGrid(gridsize);
dim3 dimBlock(blocksize);

detectConflicts<<<dimGrid, dimBlock>>>(adjacencyMatrixD, boundaryListD, colorsD, conflictD, GRAPHSIZE, boundarySize);

hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(conflict, conflictD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);

hipFree(adjacencyMatrixD);
hipFree(colorsD);
hipFree(conflictD);
hipFree(boundaryListD);
}




// Author:Peihong & Pascal
// Description: Merging of colorConfilctDetection & subGraphColoring
// to save on data transfer time
extern "C"
void colorAndConflict(int *adjacencyMatrix, int *boundaryList, int *graphColors, int *conflict, int boundarySize, int maxDegree)
{
int *adjacencyMatrixD, *colorsD, *conflictD, *boundaryListD;
int gridsize = ceil((float)boundarySize/(float)SUBSIZE_BOUNDARY);
int blocksize = SUBSIZE_BOUNDARY;

hipEvent_t start_col, start_confl, stop_col, stop_confl, start_mem, stop_mem;
float elapsedTime_memory, elapsedTime_col, elapsedTime_confl;


// memory transfer
hipEventCreate(&start_mem);
hipEventCreate(&stop_mem);
hipEventRecord(start_mem, 0);

hipMalloc((void**)&adjacencyMatrixD, GRAPHSIZE*GRAPHSIZE*sizeof(int));
hipMalloc((void**)&colorsD, GRAPHSIZE*sizeof(int));
hipMalloc((void**)&conflictD, boundarySize*sizeof(int));
hipMalloc((void**)&boundaryListD, boundarySize*sizeof(int));

hipMemcpy(adjacencyMatrixD, adjacencyMatrix, GRAPHSIZE*GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(colorsD, graphColors, GRAPHSIZE*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(boundaryListD, boundaryList, boundarySize*sizeof(int), hipMemcpyHostToDevice);


hipEventRecord(stop_mem, 0);
hipEventSynchronize(stop_mem);




dim3 dimGrid_col(GRIDSIZE);
dim3 dimBlock_col(BLOCKSIZE);

dim3 dimGrid_confl(gridsize);
dim3 dimBlock_confl(blocksize);


// Graph coloring
hipEventCreate(&start_col);
hipEventCreate(&stop_col);
hipEventRecord(start_col, 0);

colorGraph<<<dimGrid_col, dimBlock_col>>>(adjacencyMatrixD, colorsD, GRAPHSIZE, maxDegree);


hipEventRecord(stop_col, 0);
hipEventSynchronize(stop_col);



// Conflict resolution
hipEventCreate(&start_confl);
hipEventCreate(&stop_confl);
hipEventRecord(start_confl, 0);

detectConflicts<<<dimGrid_confl, dimBlock_confl>>>(adjacencyMatrixD, boundaryListD, colorsD, conflictD, GRAPHSIZE, boundarySize);

hipEventRecord(stop_confl, 0);
hipEventSynchronize(stop_confl);

hipEventElapsedTime(&elapsedTime_memory, start_mem, stop_mem);
hipEventElapsedTime(&elapsedTime_col, start_col, stop_col);
hipEventElapsedTime(&elapsedTime_confl, start_confl, stop_confl);
cout << "GPU time ~ Memory: " << elapsedTime_memory << " Color: " << elapsedTime_col << " Conflict: " << elapsedTime_confl << endl;


hipMemcpy(graphColors, colorsD, GRAPHSIZE*sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(conflict, conflictD, boundarySize*sizeof(int), hipMemcpyDeviceToHost);

hipFree(adjacencyMatrixD);
hipFree(colorsD);
hipFree(conflictD);
hipFree(boundaryListD);
}
